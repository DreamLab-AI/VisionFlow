#include <hip/hip_runtime.h>

extern "C" {
    // This struct matches the Rust BinaryNodeData struct
    struct BinaryNodeData {
        float position[3];    // 12 bytes - matches Rust [f32; 3]
        float velocity[3];    // 12 bytes - matches Rust [f32; 3]
        // These fields are used internally but not transmitted over the wire
        // The binary_protocol.rs sets default values when decoding
        unsigned char mass;   // 1 byte - matches Rust u8
        unsigned char flags;  // 1 byte - matches Rust u8
        unsigned char padding[2]; // 2 bytes - matches Rust padding
    };

    __global__ void compute_forces_kernel(
        BinaryNodeData* nodes,
        int num_nodes,
        float spring_k,
        float damping,
        float repel_k,
        float dt,
        float max_repulsion_dist,
        float viewport_bounds,
        int iteration_count
    ) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= num_nodes) return;

        const float MAX_FORCE = 3.0f; // Reduced maximum force magnitude
        const float MAX_VELOCITY = 0.02f; // Stricter velocity cap to prevent momentum buildup
        const float MIN_DISTANCE = 0.15f; // Slightly increased minimum distance
        
        // Progressive force application parameters
        // First 100 iterations use a ramp-up factor
        const int WARMUP_ITERATIONS = 100;
        float ramp_up_factor = 1.0f;
        
        if (iteration_count < WARMUP_ITERATIONS) {
            // Gradually increase from 0.01 to 1.0 over WARMUP_ITERATIONS
            ramp_up_factor = 0.01f + (iteration_count / (float)WARMUP_ITERATIONS) * 0.99f;
            
            // Also use higher damping in initial iterations to stabilize the system
            damping = fmaxf(damping, 0.9f - 0.4f * (iteration_count / (float)WARMUP_ITERATIONS));
        }
        
        float3 total_force = make_float3(0.0f, 0.0f, 0.0f);
        float3 pos = make_float3(nodes[idx].position[0], nodes[idx].position[1], nodes[idx].position[2]);
        float3 vel = make_float3(nodes[idx].velocity[0], nodes[idx].velocity[1], nodes[idx].velocity[2]);

        // Zero out velocity in the very first iterations to prevent explosion
        if (iteration_count < 5) {
            vel = make_float3(0.0f, 0.0f, 0.0f);
        }
        
        // Convert mass from u8 to float (approximately 0-1 range)
        float mass;
        if (nodes[idx].mass == 0) {
            mass = 0.5f; // Default mid-range mass value
        } else {
            mass = (nodes[idx].mass + 1.0f) / 256.0f; // Add 1 to avoid zero mass
        }
        
        bool is_active = true; // All nodes are active by default
        
        if (!is_active) return; // Skip inactive nodes
        
        // Process all node interactions
        for (int j = 0; j < num_nodes; j++) {
            if (j == idx) continue;
            
            // All nodes are considered active by default
            // We no longer check the flags since all nodes are treated as active
            
            // Handle other node's mass the same way
            float other_mass = (nodes[j].mass == 0) ? 0.5f : (nodes[j].mass + 1.0f) / 256.0f;
            
            float3 other_pos = make_float3(
                nodes[j].position[0],
                nodes[j].position[1],
                nodes[j].position[2]
            );
            
            float3 diff = make_float3(
                other_pos.x - pos.x,
                other_pos.y - pos.y,
                other_pos.z - pos.z
            );
            
            float dist = sqrtf(diff.x * diff.x + diff.y * diff.y + diff.z * diff.z);
            // Only process if nodes are at a meaningful distance apart
            if (dist > MIN_DISTANCE) {
                float3 dir = make_float3(
                    diff.x / dist,
                    diff.y / dist,
                    diff.z / dist
                );
                
                // Apply spring forces to all nodes by default
                {
                    // Use natural length of 1.0 to match world units
                    float natural_length = 1.0f;
                    
                    // Progressive spring forces - stronger when further apart
                    // Apply the ramp_up_factor to gradually increase spring forces
                    float spring_force = spring_k * ramp_up_factor * (dist - natural_length);
                    
                    // Apply progressively stronger springs for very distant nodes
                    if (dist > natural_length * 3.0f) {
                        spring_force *= (1.0f + (dist - natural_length * 3.0f) * 0.1f);
                    }
                    
                    
                    float spring_scale = mass * other_mass;
                    float force_magnitude = spring_force * spring_scale;
                    
                    // Repulsion forces - only apply at close distances
                    if (dist < max_repulsion_dist) {
                        float repel_scale = repel_k * mass * other_mass;
                        // Apply the ramp_up_factor to gradually increase repulsion forces
                        float dist_sq = fmaxf(dist * dist, MIN_DISTANCE);
                        // Cap maximum repulsion force to prevent explosion
                        float repel_force = fminf(repel_scale / dist_sq, repel_scale * 2.0f);
                        total_force.x -= dir.x * repel_force;
                        total_force.y -= dir.y * repel_force;
                        total_force.z -= dir.z * repel_force;
                    } else {
                        // Always apply spring forces
                        total_force.x += dir.x * force_magnitude;
                        total_force.y += dir.y * force_magnitude;
                        total_force.z += dir.z * force_magnitude;
                    }
                }
            }
        }
        
        // Stronger center gravity to prevent nodes from drifting too far
        float center_strength = 0.015f * mass * ramp_up_factor; // Apply ramp_up to center gravity too
        float center_dist = sqrtf(pos.x*pos.x + pos.y*pos.y + pos.z*pos.z);
        if (center_dist > 3.0f) { // Apply at shorter distances
            float center_factor = center_strength * (center_dist - 3.0f) / center_dist;
            total_force.x -= pos.x * center_factor;
            total_force.y -= pos.y * center_factor;
            total_force.z -= pos.z * center_factor;
        }

        // Calculate total force magnitude
        float force_magnitude = sqrtf(
            total_force.x*total_force.x + 
            total_force.y*total_force.y + 
            total_force.z*total_force.z);
        
        // Scale down excessive forces to prevent explosion
        if (force_magnitude > MAX_FORCE) {
            float scale_factor = MAX_FORCE / force_magnitude;
            total_force.x *= scale_factor;
            total_force.y *= scale_factor;
            total_force.z *= scale_factor;
            
            // Additional logging to help debug extreme forces after randomization
            if (idx == 0 && iteration_count < 5)
                printf("Force clamped from %f to %f (iteration %d)\n", force_magnitude, MAX_FORCE, iteration_count);
        }

        // Apply damping and bounded forces to velocity
        vel.x = vel.x * (1.0f - damping) + fminf(MAX_FORCE, fmaxf(-MAX_FORCE, total_force.x)) * dt;
        vel.y = vel.y * (1.0f - damping) + fminf(MAX_FORCE, fmaxf(-MAX_FORCE, total_force.y)) * dt;
        vel.z = vel.z * (1.0f - damping) + fminf(MAX_FORCE, fmaxf(-MAX_FORCE, total_force.z)) * dt;
        
        // Apply STRICT velocity cap to prevent runaway momentum
        float vel_magnitude = sqrtf(vel.x*vel.x + vel.y*vel.y + vel.z*vel.z);
        if (vel_magnitude > MAX_VELOCITY) {
            float scale_factor = MAX_VELOCITY / vel_magnitude;
            vel.x *= scale_factor;
            vel.y *= scale_factor;
            vel.z *= scale_factor;
        }
        
        // Update position
        pos.x += vel.x * dt;
        pos.y += vel.y * dt;
        pos.z += vel.z * dt;

        // Progressive boundary approach - stronger the further you go
        if (viewport_bounds > 0.0f && iteration_count > 10) { // Only apply boundary after initial stabilization
            float soft_margin = 0.3f * viewport_bounds; // 30% soft boundary
            float bound_with_margin = viewport_bounds - soft_margin;

            // Apply progressively stronger boundary forces
            if (fabsf(pos.x) > bound_with_margin) {
                pos.x *= 0.92f; // Pull back by 8%
                // Also add dampening to velocity in this direction
                vel.x *= 0.85f;
            }
            if (fabsf(pos.y) > bound_with_margin) {
                pos.y *= 0.92f; // Pull back by 8%
                vel.y *= 0.85f;
            }
            if (fabsf(pos.z) > bound_with_margin) {
                pos.z *= 0.92f; // Pull back by 8%
                vel.z *= 0.85f;
            }
        }

        // Store results back
        nodes[idx].position[0] = pos.x;
        nodes[idx].position[1] = pos.y;
        nodes[idx].position[2] = pos.z;
        nodes[idx].velocity[0] = vel.x;
        nodes[idx].velocity[1] = vel.y;
        nodes[idx].velocity[2] = vel.z;

        // Debug output for first node
        if (idx == 0 && (iteration_count < 5 || iteration_count % 20 == 0)) {
            float force_mag = sqrtf(
                total_force.x * total_force.x +
                total_force.y * total_force.y +
                total_force.z * total_force.z
            );
            printf("Node %d: force_mag=%f, pos=(%f,%f,%f), vel=(%f,%f,%f)\n",
                idx, force_mag, 
                pos.x, pos.y, pos.z,
                vel.x, vel.y, vel.z);
                
            // More detailed logging during initialization
            if (iteration_count < WARMUP_ITERATIONS)
                printf("Node %d: iteration=%d, ramp_up=%f, damping=%f\n", idx, iteration_count, ramp_up_factor, damping);
        }
    }
}
